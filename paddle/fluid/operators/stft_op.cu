#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/operators/conj_op.h"
#include "paddle/fluid/operators/eigen/eigen_function.h"
#include "paddle/fluid/operators/stft_op.h"
#include "paddle/fluid/operators/transpose_op.h"
#include "paddle/fluid/platform/dynload/hipfft/hipfft.h"
#include "paddle/fluid/platform/enforce.h"
#include "paddle/fluid/platform/for_range.h"

namespace paddle {
namespace operators {

class CuFFTHandle {
  ::hipfftHandle handle_;

 public:
  CuFFTHandle() {
    PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipfftCreate(&handle_));
  }

  CuFFTHandle(const CuFFTHandle& other) = delete;
  CuFFTHandle& operator=(const CuFFTHandle& other) = delete;

  CuFFTHandle(CuFFTHandle&& other) = delete;
  CuFFTHandle& operator=(CuFFTHandle&& other) = delete;

  ::hipfftHandle& get() { return handle_; }
  const ::hipfftHandle& get() const { return handle_; }

  ~CuFFTHandle() {
    PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipfftDestroy(handle_));
  }
};

// Create transform type enum from bools representing if input and output are
// complex
inline FFTTransformType GetFFTTransformType(
    framework::proto::VarType::Type input_dtype,
    framework::proto::VarType::Type output_dtype) {
  auto complex_input = framework::IsComplexType(input_dtype);
  auto complex_output = framework::IsComplexType(output_dtype);
  if (complex_input && complex_output) {
    return FFTTransformType::C2C;
  } else if (complex_input && !complex_output) {
    return FFTTransformType::C2R;
  } else if (!complex_input && complex_output) {
    return FFTTransformType::R2C;
  }
  PADDLE_THROW(
      platform::errors::InvalidArgument("Real to real FFTs are not supported"));
}

using ScalarType = framework::proto::VarType::Type;
const int64_t kMaxFFTNdim = 3;
const int64_t kMaxDataNdim = kMaxFFTNdim + 1;
// This struct is used to easily compute hashes of the
// parameters. It will be the **key** to the plan cache.
struct FFTConfigKey {
  // between 1 and kMaxFFTNdim, i.e., 1 <= signal_ndim <= 3
  int64_t signal_ndim_;
  // These include additional batch dimension as well.
  int64_t sizes_[kMaxDataNdim];
  int64_t input_shape_[kMaxDataNdim];
  int64_t output_shape_[kMaxDataNdim];
  FFTTransformType fft_type_;
  ScalarType value_type_;

  FFTConfigKey() = default;

  FFTConfigKey(const std::vector<int64_t>& in_shape,
               const std::vector<int64_t>& out_shape,
               const std::vector<int64_t>& signal_size,
               FFTTransformType fft_type, ScalarType value_type) {
    // Padding bits must be zeroed for hashing
    memset(this, 0, sizeof(*this));
    signal_ndim_ = signal_size.size() - 1;
    fft_type_ = fft_type;
    value_type_ = value_type;

    std::copy(signal_size.cbegin(), signal_size.cend(), sizes_);
    std::copy(in_shape.cbegin(), in_shape.cend(), input_shape_);
    std::copy(out_shape.cbegin(), out_shape.cend(), output_shape_);
  }
};

using plan_size_type = long long int;  // NOLINT
// This class contains all the information needed to execute a cuFFT plan:
//   1. the plan
//   2. the workspace size needed
class FFTConfig {
 public:
  // Only move semantics is enought for this class. Although we already use
  // unique_ptr for the plan, still remove copy constructor and assignment op so
  // we don't accidentally copy and take perf hit.
  explicit FFTConfig(const FFTConfigKey& plan_key)
      : FFTConfig(
            std::vector<int64_t>(plan_key.sizes_,
                                 plan_key.sizes_ + plan_key.signal_ndim_ + 1),
            plan_key.signal_ndim_, plan_key.fft_type_, plan_key.value_type_) {}

  // sizes are full signal, including batch size and always two-sided
  FFTConfig(const std::vector<int64_t>& sizes, const int64_t signal_ndim,
            FFTTransformType fft_type, ScalarType dtype)
      : fft_type_(fft_type), value_type_(dtype) {
    // signal sizes (excluding batch dim)
    std::vector<plan_size_type> signal_sizes(sizes.begin() + 1, sizes.end());

    // input batch size
    const auto batch = static_cast<plan_size_type>(sizes[0]);
    // const int64_t signal_ndim = sizes.size() - 1;
    PADDLE_ENFORCE_EQ(signal_ndim, sizes.size() - 1,
                      platform::errors::InvalidArgument(
                          "The signal_ndim must be equal to sizes.size() - 1,"
                          "But signal_ndim is: [%d], sizes.size() - 1 is: [%d]",
                          signal_ndim, sizes.size() - 1));

    hipDataType itype, otype, exec_type;
    const auto complex_input = has_complex_input(fft_type);
    const auto complex_output = has_complex_output(fft_type);
    if (dtype == framework::proto::VarType::FP32) {
      itype = complex_input ? HIP_C_32F : HIP_R_32F;
      otype = complex_output ? HIP_C_32F : HIP_R_32F;
      exec_type = HIP_C_32F;
    } else if (dtype == framework::proto::VarType::FP64) {
      itype = complex_input ? HIP_C_64F : HIP_R_64F;
      otype = complex_output ? HIP_C_64F : HIP_R_64F;
      exec_type = HIP_C_64F;
    } else if (dtype == framework::proto::VarType::FP16) {
      itype = complex_input ? HIP_C_16F : HIP_R_16F;
      otype = complex_output ? HIP_C_16F : HIP_R_16F;
      exec_type = HIP_C_16F;
    } else {
      PADDLE_THROW(platform::errors::InvalidArgument(
          "cuFFT only support transforms of type float16, float32 and "
          "float64"));
    }

    // disable auto allocation of workspace to use allocator from the framework
    PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipfftSetAutoAllocation(
        plan(), /* autoAllocate */ 0));

    size_t ws_size_t;

    PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipfftXtMakePlanMany(
        plan(), signal_ndim, signal_sizes.data(),
        /* inembed */ nullptr, /* base_istride */ 1, /* idist */ 1, itype,
        /* onembed */ nullptr, /* base_ostride */ 1, /* odist */ 1, otype,
        batch, &ws_size_t, exec_type));

    ws_size = ws_size_t;
  }

  FFTConfig(const FFTConfig& other) = delete;
  FFTConfig& operator=(const FFTConfig& other) = delete;

  FFTConfig(FFTConfig&& other) = delete;
  FFTConfig& operator=(FFTConfig&& other) = delete;

  const hipfftHandle& plan() const { return plan_ptr.get(); }

  FFTTransformType transform_type() const { return fft_type_; }
  ScalarType data_type() const { return value_type_; }
  size_t workspace_size() const { return ws_size; }

 private:
  CuFFTHandle plan_ptr;
  size_t ws_size;
  FFTTransformType fft_type_;
  ScalarType value_type_;
};

template <typename Key>
struct KeyHash {
  // Key must be a POD because we read out its memory
  // contenst as char* when hashing
  static_assert(std::is_pod<Key>::value, "Key must be plain old data type");

  size_t operator()(const Key& params) const {
    auto ptr = reinterpret_cast<const uint8_t*>(&params);
    uint32_t value = 0x811C9DC5;
    for (int i = 0; i < static_cast<int>(sizeof(Key)); ++i) {
      value ^= ptr[i];
      value *= 0x01000193;
    }
    return static_cast<size_t>(value);
  }
};

template <typename Key>
struct KeyEqual {
  // Key must be a POD because we read out its memory
  // contenst as char* when comparing
  static_assert(std::is_pod<Key>::value, "Key must be plain old data type");

  bool operator()(const Key& a, const Key& b) const {
    auto ptr1 = reinterpret_cast<const uint8_t*>(&a);
    auto ptr2 = reinterpret_cast<const uint8_t*>(&b);
    return memcmp(ptr1, ptr2, sizeof(Key)) == 0;
  }
};

constexpr size_t CUFFT_MAX_PLAN_NUM = std::numeric_limits<size_t>::max();
constexpr size_t CUFFT_DEFAULT_CACHE_SIZE = 4096;
class FFTConfigCache {
 public:
  using kv_t = typename std::pair<FFTConfigKey, FFTConfig>;
  using map_t = typename std::unordered_map<
      std::reference_wrapper<FFTConfigKey>, typename std::list<kv_t>::iterator,
      KeyHash<FFTConfigKey>, KeyEqual<FFTConfigKey>>;
  using map_kkv_iter_t = typename map_t::iterator;

  FFTConfigCache() : FFTConfigCache(CUFFT_DEFAULT_CACHE_SIZE) {}

  explicit FFTConfigCache(int64_t max_size) { _set_max_size(max_size); }

  FFTConfigCache(const FFTConfigCache& other) = delete;
  FFTConfigCache& operator=(const FFTConfigCache& other) = delete;

  FFTConfigCache(FFTConfigCache&& other) noexcept
      : _usage_list(std::move(other._usage_list)),
        _cache_map(std::move(other._cache_map)),
        _max_size(other._max_size) {}

  FFTConfigCache& operator=(FFTConfigCache&& other) noexcept {
    _usage_list = std::move(other._usage_list);
    _cache_map = std::move(other._cache_map);
    _max_size = other._max_size;
    return *this;
  }

  // If key is in this cache, return the cached config. Otherwise, emplace the
  // config in this cache and return it.
  FFTConfig& lookup(FFTConfigKey params) {
    PADDLE_ENFORCE_GT(_max_size, 0,
                      platform::errors::InvalidArgument(
                          "The max size of FFTConfigCache must be great than 0,"
                          "But received is [%d]",
                          _max_size));

    map_kkv_iter_t map_it = _cache_map.find(params);
    // Hit, put to list front
    if (map_it != _cache_map.end()) {
      _usage_list.splice(_usage_list.begin(), _usage_list, map_it->second);
      return map_it->second->second;
    }

    // Miss
    // remove if needed
    if (_usage_list.size() >= _max_size) {
      auto last = _usage_list.end();
      last--;
      _cache_map.erase(last->first);
      _usage_list.pop_back();
    }

    // construct new plan at list front, then insert into _cache_map
    _usage_list.emplace_front(std::piecewise_construct,
                              std::forward_as_tuple(params),
                              std::forward_as_tuple(params));
    auto kv_it = _usage_list.begin();
    _cache_map.emplace(std::piecewise_construct,
                       std::forward_as_tuple(kv_it->first),
                       std::forward_as_tuple(kv_it));
    return kv_it->second;
  }

  void clear() {
    _cache_map.clear();
    _usage_list.clear();
  }

  void resize(int64_t new_size) {
    _set_max_size(new_size);
    auto cur_size = _usage_list.size();
    if (cur_size > _max_size) {
      auto delete_it = _usage_list.end();
      for (size_t i = 0; i < cur_size - _max_size; i++) {
        delete_it--;
        _cache_map.erase(delete_it->first);
      }
      _usage_list.erase(delete_it, _usage_list.end());
    }
  }

  size_t size() const { return _cache_map.size(); }

  size_t max_size() const noexcept { return _max_size; }

  std::mutex mutex;

 private:
  // Only sets size and does value check. Does not resize the data structures.
  void _set_max_size(int64_t new_size) {
    // We check that 0 <= new_size <= CUFFT_MAX_PLAN_NUM here. Since
    // CUFFT_MAX_PLAN_NUM is of type size_t, we need to do non-negativity check
    // first.
    PADDLE_ENFORCE_GE(
        new_size, 0,
        platform::errors::InvalidArgument(
            "cuFFT plan cache size must be non-negative, But received is [%d]",
            new_size));
    PADDLE_ENFORCE_LE(new_size, CUFFT_MAX_PLAN_NUM,
                      platform::errors::InvalidArgument(
                          "cuFFT plan cache size can not be larger than [%d], "
                          "But received is [%d]",
                          CUFFT_MAX_PLAN_NUM, new_size));
    _max_size = static_cast<size_t>(new_size);
  }

  std::list<kv_t> _usage_list;
  map_t _cache_map;
  size_t _max_size;
};

// Calculates the normalization constant
double fft_normalization_scale(FFTNormMode normalization,
                               const std::vector<int64_t>& sizes,
                               const std::vector<int64_t>& dims) {
  // auto norm = static_cast<fft_norm_mode>(normalization);
  if (normalization == FFTNormMode::none) {
    return static_cast<double>(1.0);
  }

  int64_t signal_numel = 1;
  for (auto dim : dims) {
    signal_numel *= sizes[dim];
  }
  const double scale_denom = (normalization == FFTNormMode::by_sqrt_n)
                                 ? std::sqrt(signal_numel)
                                 : static_cast<double>(signal_numel);
  return static_cast<double>(1.0 / scale_denom);
}

template <typename DeviceContext, typename T>
void exec_normalization(const DeviceContext& ctx, const Tensor* in, Tensor* out,
                        FFTNormMode normalization,
                        const std::vector<int64_t>& sizes,
                        const std::vector<int64_t>& axes) {
  double scale = fft_normalization_scale(normalization, sizes, axes);
  if (scale != 1.0) {
    auto eigen_out = framework::EigenVector<T>::Flatten(*out);
    auto eigen_in = framework::EigenVector<T>::Flatten(*in);
    auto dev = ctx.eigen_device();
    EigenScale<Eigen::GpuDevice, T>::Eval(*dev, eigen_out, eigen_in,
                                          static_cast<T>(scale),
                                          static_cast<T>(0), false);
  } else {
    framework::TensorCopy(*in, ctx.GetPlace(), out);
  }
}

FFTConfigKey create_fft_configkey(const framework::Tensor& input,
                                  const framework::Tensor& output,
                                  int signal_ndim) {
  // Create the transform plan (either from cache or locally)
  const auto value_type = framework::IsComplexType(input.type())
                              ? framework::ToRealType(input.type())
                              : input.type();
  auto fft_type = GetFFTTransformType(input.type(), output.type());
  // signal sizes
  std::vector<int64_t> signal_size(signal_ndim + 1);

  signal_size[0] = input.dims()[0];
  for (int64_t i = 1; i <= signal_ndim; ++i) {
    auto in_size = input.dims()[i];
    auto out_size = output.dims()[i];
    signal_size[i] = std::max(in_size, out_size);
  }
  FFTConfigKey key(framework::vectorize(input.dims()),
                   framework::vectorize(output.dims()), signal_size, fft_type,
                   value_type);
  return key;
}

// Execute a pre-planned transform
static void exec_cufft_plan_raw(const FFTConfig& config, void* in_data,
                                void* out_data, bool forward) {
  auto& plan = config.plan();

  PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipfftXtExec(
      plan, in_data, out_data, forward ? HIPFFT_FORWARD : HIPFFT_BACKWARD));
}

template <typename DeviceContext, typename Ti, typename To>
void exec_cufft_plan(const DeviceContext& ctx, const FFTConfig& config,
                     framework::Tensor* input, framework::Tensor* output,
                     bool forward) {
  // execute transform plan
  auto fft_type = config.transform_type();
  if (fft_type == FFTTransformType::C2R && forward) {
    forward = false;
    framework::Tensor input_conj(input->type());
    input_conj.mutable_data<Ti>(input->dims(), ctx.GetPlace());
    platform::ForRange<DeviceContext> for_range(ctx, input->numel());
    math::ConjFunctor<Ti> functor(input->data<Ti>(), input->numel(),
                                  input_conj.data<Ti>());
    for_range(functor);
    exec_cufft_plan_raw(config, input_conj.data<void>(), output->data<void>(),
                        forward);
  } else if (fft_type == FFTTransformType::R2C && !forward) {
    forward = true;
    framework::Tensor out_conj(output->type());
    out_conj.mutable_data<To>(output->dims(), ctx.GetPlace());
    exec_cufft_plan_raw(config, input->data<void>(), out_conj.data<void>(),
                        forward);

    platform::ForRange<DeviceContext> for_range(ctx, output->numel());
    math::ConjFunctor<To> functor(out_conj.data<To>(), output->numel(),
                                  output->data<To>());
    for_range(functor);
  } else {
    exec_cufft_plan_raw(config, input->data<void>(), output->data<void>(),
                        forward);
  }
}

static std::vector<std::unique_ptr<FFTConfigCache>> plan_caches;
static std::mutex plan_caches_mutex;

static inline FFTConfigCache& get_fft_plan_cache(int64_t device_index) {
  std::lock_guard<std::mutex> guard(plan_caches_mutex);

  if (device_index >= plan_caches.size()) {
    plan_caches.resize(device_index + 1);
  }

  if (!plan_caches[device_index]) {
    plan_caches[device_index] = std::make_unique<FFTConfigCache>();
  }

  return *plan_caches[device_index];
}

// Execute a general unnormalized fft operation (can be c2c, onesided r2c or
// onesided c2r)
template <typename DeviceContext, typename Ti, typename To>
void exec_fft(const DeviceContext& ctx, const Tensor* X, Tensor* out,
              const std::vector<int64_t>& dim, bool forward) {
  const auto x_dims = framework::vectorize(X->dims());
  const int64_t ndim = static_cast<int64_t>(X->dims().size());
  auto tensor_place = ctx.GetPlace();

  // make a dim permutation
  std::vector<int> dim_permute(ndim);
  std::iota(dim_permute.begin(), dim_permute.end(), int{0});
  std::vector<bool> is_transformed_dim(ndim);
  for (const auto& d : dim) {
    is_transformed_dim[d] = true;
  }
  auto batch_end =
      std::partition(dim_permute.begin(), dim_permute.end(),
                     [&](int64_t d) { return !is_transformed_dim[d]; });
  std::sort(dim_permute.begin(), batch_end);
  std::copy(dim.cbegin(), dim.cend(), batch_end);

  // transpose input according to dim permutation
  auto transposed_input_shape = X->dims().transpose(dim_permute);
  framework::Tensor transposed_input;
  transposed_input.Resize(transposed_input_shape);
  transposed_input.mutable_data<Ti>(tensor_place);
  TransCompute<DeviceContext, Ti>(ndim, ctx, *X, &transposed_input,
                                  dim_permute);

  // Reshape batch dimensions into a single dimension
  const int64_t signal_ndim = static_cast<int64_t>(dim.size());
  std::vector<int64_t> collapsed_input_shape(signal_ndim + 1);

  auto transposed_input_shape_ = framework::vectorize(transposed_input_shape);
  const int64_t batch_dims = ndim - signal_ndim;
  auto batch_size =
      std::accumulate(transposed_input_shape_.begin(),
                      transposed_input_shape_.begin() + batch_dims,
                      static_cast<int>(1), std::multiplies<int>());
  collapsed_input_shape[0] = batch_size;

  std::copy(transposed_input_shape_.begin() + batch_dims,
            transposed_input_shape_.end(), collapsed_input_shape.begin() + 1);

  framework::Tensor& collapsed_input = transposed_input;
  collapsed_input.Resize(framework::make_ddim(collapsed_input_shape));

  // make a collpased output
  const auto out_dims = framework::vectorize(out->dims());
  std::vector<int64_t> collapsed_output_shape(1 + signal_ndim);
  collapsed_output_shape[0] = batch_size;
  for (size_t i = 0; i < dim.size(); ++i) {
    collapsed_output_shape[i + 1] = out_dims[dim[i]];
  }
  framework::Tensor collapsed_output;
  collapsed_output.Resize(framework::make_ddim(collapsed_output_shape));
  collapsed_output.mutable_data<To>(tensor_place);

  FFTConfig* config = nullptr;

  std::unique_ptr<FFTConfig> config_ = nullptr;
  // create plan
  FFTConfigKey key =
      create_fft_configkey(collapsed_input, collapsed_output, signal_ndim);
  if (CUFFT_VERSION < 10200) {
    const int64_t device_id = static_cast<int64_t>(
        reinterpret_cast<const platform::CUDAPlace*>(&collapsed_input.place())
            ->GetDeviceId());
    FFTConfigCache& plan_cache = get_fft_plan_cache(device_id);
    std::unique_lock<std::mutex> guard(plan_cache.mutex, std::defer_lock);
    guard.lock();
    config = &(plan_cache.lookup(key));
  } else {
    config_ = std::make_unique<FFTConfig>(key);
    config = config_.get();
  }
  // prepare cufft for execution
  PADDLE_ENFORCE_CUDA_SUCCESS(
      platform::dynload::hipfftSetStream(config->plan(), ctx.stream()));
  framework::Tensor workspace_tensor;
  workspace_tensor.mutable_data<To>(tensor_place, config->workspace_size());
  PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipfftSetWorkArea(
      config->plan(), workspace_tensor.data<To>()));
  // execute transform plan
  exec_cufft_plan<DeviceContext, Ti, To>(ctx, *config, &collapsed_input,
                                         &collapsed_output, forward);

  // Inverting output by reshape and transpose to original batch and dimension
  auto transposed_out_shape = out->dims().transpose(dim_permute);

  collapsed_output.Resize(transposed_out_shape);
  auto& transposed_output = collapsed_output;

  std::vector<int> reverse_dim_permute(ndim);
  for (size_t i = 0; i < ndim; i++) {
    reverse_dim_permute[dim_permute[i]] = i;
  }

  TransCompute<DeviceContext, To>(ndim, ctx, transposed_output, out,
                                  reverse_dim_permute);
}

namespace {

// Use the optimized path to perform single R2C or C2R if transformation dim is
// supported by cuFFT
bool use_optimized_fft_path(const std::vector<int64_t>& axes) {
  // For performance reason, when axes starts with (0, 1), do not use the
  // optimized path.
  if (axes.size() > kMaxFFTNdim ||
      (axes.size() >= 2 && axes[0] == 0 && axes[1] == 1)) {
    return false;
  } else {
    return true;
  }
}

}  // anonymous namespace

template <typename Ti, typename To>
struct FFTC2CFunctor<platform::CUDADeviceContext, Ti, To> {
  void operator()(const platform::CUDADeviceContext& ctx, const Tensor* X,
                  Tensor* out, const std::vector<int64_t>& axes,
                  FFTNormMode normalization, bool forward) {
    if (axes.empty()) {
      framework::TensorCopy(*X, ctx.GetPlace(), out);
      return;
    }

    framework::Tensor* p_out = out;
    std::vector<int64_t> out_dims = framework::vectorize(X->dims());
    std::vector<int64_t> working_axes(axes.begin(), axes.end());
    std::vector<int64_t> first_dims;
    size_t max_dims;
    framework::Tensor working_tensor;
    working_tensor.mutable_data<Ti>(X->dims(), ctx.GetPlace());
    framework::Tensor* p_working_tensor = &working_tensor;
    framework::TensorCopy(*X, ctx.GetPlace(), &working_tensor);

    while (true) {
      max_dims =
          std::min(static_cast<size_t>(kMaxFFTNdim), working_axes.size());
      first_dims.assign(working_axes.end() - max_dims, working_axes.end());

      exec_fft<platform::CUDADeviceContext, Ti, To>(ctx, p_working_tensor,
                                                    p_out, first_dims, forward);
      working_axes.resize(working_axes.size() - max_dims);
      first_dims.clear();

      if (working_axes.empty()) {
        break;
      }

      std::swap(p_out, p_working_tensor);
    }
    exec_normalization<platform::CUDADeviceContext, To>(
        ctx, p_out, out, normalization, out_dims, axes);
  }
};

// n dimension real to complex FFT use cufft lib
template <typename Ti, typename To>
struct FFTR2CFunctor<platform::CUDADeviceContext, Ti, To> {
  void operator()(const platform::CUDADeviceContext& ctx, const Tensor* X,
                  Tensor* out, const std::vector<int64_t>& axes,
                  FFTNormMode normalization, bool forward) {
    // Step1: R2C transform on the last dimension
    framework::Tensor* r2c_out = out;
    const std::vector<int64_t> last_dim{axes.back()};
    std::vector<int64_t> out_dims = framework::vectorize(out->dims());
    exec_fft<platform::CUDADeviceContext, Ti, To>(ctx, X, r2c_out, last_dim,
                                                  forward);

    // Step2: C2C transform on the remaining dimension
    framework::Tensor c2c_out;
    if (axes.size() > 1) {
      c2c_out.mutable_data<To>(out->dims(), ctx.GetPlace());
      std::vector<int64_t> remain_dim(axes.begin(), axes.end() - 1);
      FFTC2CFunctor<platform::CUDADeviceContext, To, To> fft_c2c_func;
      fft_c2c_func(ctx, r2c_out, &c2c_out, remain_dim, FFTNormMode::none,
                   forward);
    }

    const auto in_sizes = framework::vectorize(X->dims());
    framework::Tensor* norm_tensor = axes.size() > 1 ? &c2c_out : r2c_out;
    exec_normalization<platform::CUDADeviceContext, To>(
        ctx, norm_tensor, out, normalization, in_sizes, axes);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;

REGISTER_OP_CUDA_KERNEL(
    stft, ops::StftKernel<paddle::platform::CUDADeviceContext, float>,
    ops::StftKernel<paddle::platform::CUDADeviceContext, double>);
